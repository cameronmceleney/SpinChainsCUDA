#include "hip/hip_runtime.h"
#include <vccorlib.h>
#include "Numerical_Methods_Class.cuh"

void Numerical_Methods_Class::NumericalMethodsMain() {

    NumericalMethodsFlags();
    NumericalMethodsParameters();
    NumericalMethodsProcessing();

    // ###################### Core Method Invocations ######################
    // Order is intentional, and must be maintained!
    FinalChecks();
    SetShockwaveConditions();
    if (_useMultilayer) {SetDampingRegionMulti();} else {SetDampingRegion();}
    SetDrivingRegion();
    SetExchangeVector();
    if (!_useMultilayer) {SetInitialMagneticMoments();}
}
void Numerical_Methods_Class::NumericalMethodsFlags() {

    // Debugging Flags
    _shouldTrackMValues = true;

    // Model Type
    _useLLG = true;
    _useSLLG = false;

    // Interaction Flags
    _hasShockwave = false;
    _useDipolar = false;
    _useZeeman = true;

    // Material Flags
    _isFM = GV.GetIsFerromagnetic();
    _useMultilayer = false;

    // Drive Flags
    _centralDrive = false;
    _driveAllLayers = false;
    _dualDrive = false;
    _lhsDrive = true;
    _hasStaticDrive = false;
    _shouldDriveCease = false;

    // Output Flags
    _printAllData = false;
    _printFixedLines = true;
    _printFixedSites = false;
}
void Numerical_Methods_Class::NumericalMethodsParameters() {

    // Main Parameters
    _ambientTemperature = 273; // Kelvin
    _drivingFreq = 42.5 * 1e9;
    _dynamicBiasField = 3e-3;
    _forceStopAtIteration = -1;
    _gyroMagConst = GV.GetGyromagneticConstant();
    _maxSimTime = 0.7e-9;
    _satMag = 0.010032;
    _stepsize = 1e-15;

    // Shockwave Parameters
    _iterStartShock = 0.0;
    _iterEndShock = 0.0001;
    _shockwaveGradientTime = 1;
    _shockwaveInitialStrength = 0;  // Set equal to _dynamicBiasField if NOT starting at time=0
    _shockwaveMax = 3e-3;
    _shockwaveScaling = 1;

    // Data Output Parameters
    _fixed_output_sites = {12158, 14529, 15320};
    _numberOfDataPoints = 100; //static_cast<int>(_maxSimTime / _recordingInterval);
    _recordingInterval = 0.7e-11;
    _layerOfInterest = 1;

    // Damping Factors
    _gilbertConst  = 1e-4;
    _gilbertLower = _gilbertConst;
    _gilbertUpper = 1e0;

    // Spin chain and multi-layer Parameters
    _drivingRegionWidth = 200;
    _numberNeighbours = -1;
    _numSpinsDamped = 0;
    _totalLayers = 1;
}
void Numerical_Methods_Class::NumericalMethodsProcessing() {
    // Computations based upon other inputs
    _drivingAngFreq = 2 * M_PI * _drivingFreq;
    _muMagnitudeIron *= _bohrMagneton;  // Conversion to Am^2
    _dipoleConstant = _permFreeSpace / (4.0 * M_PI);

    _iterationEnd = static_cast<int>(_maxSimTime / _stepsize);
    _stepsizeHalf = _stepsize / 2.0;

    _layerSpinsInChain = {_drivingRegionWidth, GV.GetNumSpins()};

    if (!_useMultilayer) { _layerSpinsInChain[0] = GV.GetNumSpins(); }

    _layerSpinPairs.clear();
    _layerTotalSpins.clear();
    for (int& spinsInChain: _layerSpinsInChain) {
        _layerSpinPairs.push_back(spinsInChain - 1);
        _layerTotalSpins.push_back(spinsInChain + 2 * _numSpinsDamped);
    }
    _gilbertVectorMulti.resize(_totalLayers, {0});

    _layerOfInterest -= 1;  // To correct for 0-indexing

    _numSpinsInChain = GV.GetNumSpins();
    _numberOfSpinPairs = _numSpinsInChain - 1;
    GV.SetNumSpins(_numSpinsInChain + 2 * _numSpinsDamped);

    if (_isFM)
        _anisotropyField = 0;
    else if (!_isFM)
        _anisotropyField = GV.GetAnisotropyField();

    if (!_useZeeman)
        GV.SetStaticBiasField(0);
}

void Numerical_Methods_Class::FinalChecks() {

    if (_shouldDriveCease and _iterEndShock <= 0) {
        std::cout << "Warning: [_shouldDriveCease: True] however [_iterEndShock: " << _iterEndShock << " ! > 0.0]"
                  << std::endl;
        exit(1);
    }

    if (_hasShockwave and _iterStartShock < 0) {
        std::cout << "Warning: [_hasShockwave: True] however [_iterStartShock: " << _iterStartShock << " ! > 0.0]"
                  << std::endl;
        exit(1);
    }

    if ((_printFixedSites and _printFixedLines) or (_printFixedSites and _printAllData) or
        (_printFixedLines and _printAllData)) {
        std::cout << "Warning: Multiple output flags detected. [_printFixedSites: " << _printFixedSites
                  << "] | [_printFixedLines: " << _printFixedLines << "] | [_printAllData: " << _printAllData << "]"
                  << std::endl;
        exit(1);
    }

    if ((_lhsDrive && _centralDrive) || (_lhsDrive && _dualDrive) || (_centralDrive && _dualDrive)) {
        std::cout << "Warning: two (or more) conflicting driving region booleans were TRUE"
                  << "\n_lhsDrive: " << _lhsDrive << "\n_centralDrive: " << _centralDrive << "\n_dualDrive: " << _dualDrive
                  << "\n\nExiting...";
        exit(1);
    }

    if (_printFixedSites and _fixed_output_sites.empty()) {
        std::cout << "Warning: Request to print fixed sites, but no sites were given [_fixed_output_sites: (";
        for (int & fixed_out_val : _fixed_output_sites)
                std::cout << fixed_out_val << ", ";
        std::cout << ")].";
        exit(1);
    }

    if (_numberOfDataPoints > _iterationEnd) {
        std::cout << "Warning: You tried to print more data than was generated [_numberOfDataPoints > _iterationEnd]";
        exit(1);
    }

    if (_useLLG and _useSLLG) {
        std::cout << "Warning: You cannot use both the LLG and sLLG equations. Please choose one or the other.";
        exit(1);
    }

    if (_useMultilayer and _totalLayers < 2) {
        std::cout << "Warning: You cannot use the multilayer solver with less than 2 layers.";
        exit(1);
    }
}
void Numerical_Methods_Class::SetDrivingRegion() {
    /**
     * Set up driving regions for the system. The LHS option is solely for drives from the left of the system. The RHS options contains the
     * drive from the right, as well as an option to drive from the centre.
     */

    if (_centralDrive) {
        _drivingRegionLHS = (_numSpinsInChain/2) +_numSpinsDamped - (_drivingRegionWidth / 2);
        _drivingRegionRHS = (_numSpinsInChain/2) +_numSpinsDamped + (_drivingRegionWidth / 2);
        return;
    }

    if (_dualDrive) {
        return;
    }

    if (_lhsDrive) {
        // The +1/-1 offset excludes the zeroth spin while retaining the correct driving width
        _drivingRegionLHS = _numSpinsDamped + 1;
        _drivingRegionRHS = _drivingRegionLHS + _drivingRegionWidth - 1;
        return;
    }

    if (!_lhsDrive) {
        // The +1 is to correct the offset of adding a zeroth spin
        _drivingRegionRHS = GV.GetNumSpins() - _numSpinsDamped - 1;
        _drivingRegionLHS = _drivingRegionRHS - _drivingRegionWidth + 1;
        return;
    }

}
void Numerical_Methods_Class::SetExchangeVector() {
    /*
     * Create the arrays which house the exchange integral values. There are options to have a non-uniform exchange coded in, as well as the option to
     * induce a 'kick' into the system by initialising certain spins to have differing parameters to their neighbours.
     */
    LinspaceClass SpinChainExchange;

    if (_numSpinsDamped > 0) {
        SpinChainExchange.set_values(GV.GetExchangeMinVal(), GV.GetExchangeMaxVal(), _numberOfSpinPairs, true, false);
        _exchangeVec = SpinChainExchange.generate_array();

        std::vector<double> dampingRegionLeftExchange(_numSpinsDamped, GV.GetExchangeMinVal()), dampingRegionRightExchange(_numSpinsDamped, GV.GetExchangeMaxVal());
        dampingRegionLeftExchange.insert(dampingRegionLeftExchange.begin(), 0);
        dampingRegionRightExchange.push_back(0);

        _exchangeVec.insert(_exchangeVec.begin(), dampingRegionLeftExchange.begin(), dampingRegionLeftExchange.end());
        _exchangeVec.insert(_exchangeVec.end(), dampingRegionRightExchange.begin(), dampingRegionRightExchange.end());
    } else {
        // The linearly spaced vector is saved as the class member '_exchangeVec' simply to increase code readability
        SpinChainExchange.set_values(GV.GetExchangeMinVal(), GV.GetExchangeMaxVal(), _numberOfSpinPairs, true, true);
        _exchangeVec = SpinChainExchange.generate_array();
    }
}
void Numerical_Methods_Class::SetShockwaveConditions() {

    if (_hasShockwave) {
        _shockwaveStepsize = (_shockwaveMax - _shockwaveInitialStrength) / _shockwaveGradientTime;
    } else {
        // Ensures, on the output file, all parameter read as zero; reduces confusion when no shockwave is applied.
        _iterStartShock = 0;
        _shockwaveScaling = 0;
        _shockwaveGradientTime = 0;
        _shockwaveInitialStrength = 0;
        _shockwaveMax = _shockwaveInitialStrength * _shockwaveScaling;
        _shockwaveStepsize = (_shockwaveMax - _shockwaveInitialStrength) / _shockwaveGradientTime;
    }
}

void Numerical_Methods_Class::SetDampingRegionMulti() {
    // Generate the damping regions that are appended to either end of the spin chain.

    LinspaceClass DampingRegionLeft;
    LinspaceClass DampingRegionRight;

    if (_numSpinsDamped < 0) {
        // Guard clause.
        std::cout << "numGilbert is less than zero!";
        exit(0);
    }

    for (int i = 0; i < _totalLayers; i++) {
        std::vector<double> gilbertChain(_layerSpinsInChain[i], _gilbertConst);

        DampingRegionLeft.set_values(_gilbertUpper, _gilbertLower, _numSpinsDamped, true, false);
        DampingRegionRight.set_values(_gilbertLower, _gilbertUpper, _numSpinsDamped, true, false);
        std::vector<double> tempGilbertLHS = DampingRegionLeft.generate_array();
        std::vector<double> tempGilbertRHS = DampingRegionRight.generate_array();

        // Combine all damped regions to form vector which describes the entire spinchain.
        _gilbertVectorMulti[i].insert(_gilbertVectorMulti[i].end(), tempGilbertLHS.begin(), tempGilbertLHS.end());
        _gilbertVectorMulti[i].insert(_gilbertVectorMulti[i].end(), gilbertChain.begin(), gilbertChain.end());
        _gilbertVectorMulti[i].insert(_gilbertVectorMulti[i].end(), tempGilbertRHS.begin(), tempGilbertRHS.end());
        _gilbertVectorMulti[i].push_back(0);

        //PrintVector(_gilbertVectorMulti[i], false);
    }
}
void Numerical_Methods_Class::SetInitialMagneticMomentsMultilayer(std::vector<std::vector<std::vector<double>>>& nestedNestedVector,
                                                                  int layer, double mxInit, double myInit, double mzInit) {

    // mxInitCond[0] = _mxInit; // Only perturb initial spin

    /*
    for (int i = 0; i < GV.GetNumSpins(); i++) {
        mxInitCond[i] = 0.003162277;
        // myInitCond[i] = 0.0;
        mzInitCond[i] = 0.999994999;
    }
    */

    for (int i = 0; i < _layerTotalSpins[layer]; i++) {
        nestedNestedVector[layer].push_back({mxInit, myInit, mzInit});
    }

    // This zero is the (N+1)th spin on the RHS of the chain
    nestedNestedVector[layer].push_back({0.0, 0.0, 0.0});

}
std::vector<std::vector<std::vector<double>>> Numerical_Methods_Class::initializeNestedNestedVector(int numLayers, bool includeEnd) {
    /* Legacy code, not used in current implementation. Example implementation is below

    std::map<std::string, std::vector<std::vector<std::vector<double>>>> mValsNested3;
    mValsNested3["nestedNestedVector3"] = initializeNestedNestedVector(1, true);
    std::vector<std::vector<std::vector<double>>> m2Nest = mValsNested3["nestedNestedVector3"];
    SetInitialMagneticMomentsMultilayer(m2Nest, 1, 0, 0 , 0);
    */
    std::vector<std::vector<std::vector<double>>> innerNestedVector;
    for (int j = 0; j < numLayers; j++) {
        std::vector<std::vector<double>> innerVector;
        std::vector<double> innermostVector = {0.0, 0.0, 0.0};
        innerVector.push_back(innermostVector);
        innerNestedVector.push_back(innerVector);
    }
    return innerNestedVector;
}
std::vector<std::vector<std::vector<double>>> Numerical_Methods_Class::InitialiseNestedVectors(int& totalLayer, double& mxInit, double& myInit, double& mzInit) {

    // Initialise mapping
    std::map<std::string, std::vector<std::vector<std::vector<double>>>> mTermsMapping;

    // This is likely a very slow way to initialise (push_back is slow), but this works for now. Fix if it is a bottleneck later
    std::vector<std::vector<std::vector<double>>> innerNestedVector;
    for (int i = 0; i < totalLayer; i++) {
        std::vector<std::vector<double>> innerVector;
        std::vector<double> innermostVector = {0.0, 0.0, 0.0};
        innerVector.push_back(innermostVector);
        innerNestedVector.push_back(innerVector);
    }
    // Assign name to nested-nested vector
    mTermsMapping["nestedVector"] = innerNestedVector;

    // Assign key of map to multi-dim vector
    std::vector<std::vector<std::vector<double>>> mTermsNested = mTermsMapping["nestedVector"];

    // Invoke method to set initial magnetic moments. To call: mValsNest[layer][site][component]
    for (int layer = 0; layer < totalLayer; layer++)
        SetInitialMagneticMomentsMultilayer(mTermsNested, layer, mxInit, myInit , mzInit);

    return mTermsNested;
}

std::vector<double> Numerical_Methods_Class::DipolarInteractionIntralayer(std::vector<std::vector<double>>& mTerms,
                                                                          int& currentSite, const int& currentLayer,
                                                                          const double& exchangeStiffness) {
    /* This function calculates the dipolar interaction between the current site and its neighbours within a single layer.
     *
     * WARNING. This function assumes that every site is aligned along the x-axis which is only valid for specific
     * spin chains. This function will need to be modified to account for arbitrary spin chains.
     *
     */
    std::vector<double> totalDipoleTerms = {0.0, 0.0, 0.0};

    int vecLength, originIndex;
    if (_numberNeighbours == 0) {
        // Guard clause to ensure that the number of neighbours is not zero
        return totalDipoleTerms;
    } else if (_numberNeighbours < 0) {
        vecLength = _layerSpinsInChain[currentLayer];
        originIndex = currentSite - _numSpinsDamped - 1;
    } else {
        vecLength = 2 * _numberNeighbours + 1;
        originIndex = vecLength / 2 + 1;
    }

    if (vecLength < 0)
        std::cout << "Error: vecLength is less than zero" << std::endl;

    // Could combine these to be a single vector for memory improvements
    std::vector<double> mxTerms(vecLength, 0);
    std::vector<double> myTerms(vecLength, 0);
    std::vector<double> mzTerms(vecLength, 0);
    std::vector<int> sitePositions(vecLength, 0);

    /* This IF statement will be optimised away by passing an array of the form [x1,x2,...,y1,y2,...,z1,z2,...] when
     * CUDA is implemented; instead of giving a general 2D mTerms vector and then forcing this function to flatten.
     */
    int iFV = 0; // index flat vector
    if (_numberNeighbours < 0) {
        for (int site = _numSpinsDamped + 1; site <= vecLength + _numSpinsDamped; site++) {
            // Flatting the vectors
            mxTerms[iFV] = mTerms[site][0] * _muMagnitudeIron;
            myTerms[iFV] = mTerms[site][1] * _muMagnitudeIron;
            mzTerms[iFV] = mTerms[site][2] * _muMagnitudeIron;
            sitePositions[iFV] = site;
            iFV++;
    }
    } else {
        for (int site = currentSite - _numberNeighbours; site <= currentSite + _numberNeighbours; site++) {
            if (site < _numSpinsDamped or site >= _layerSpinsInChain[currentLayer] + _numSpinsDamped) {
                // Guard clause to skip trying assignment of any element when the index is negative
                continue;
            }
            // Flatting the vectors
            mxTerms[iFV] = mTerms[site][0] * _muMagnitudeIron;
            myTerms[iFV] = mTerms[site][1] * _muMagnitudeIron;
            mzTerms[iFV] = mTerms[site][2] * _muMagnitudeIron;
            sitePositions[iFV] = site;
            iFV++;
        }
    }
    // Here to improve readability; could be removed to improve performance
    std::vector<double> originSite = {mxTerms[originIndex], myTerms[originIndex], mzTerms[originIndex]};

    // Start of the loop over the neighbours
    for (int i = 0; i < vecLength; i++) {
        if (i == originIndex) {
            // Guard clause to ensure that the origin site is not included in the calculation
            continue;
        }

        // Moment at site i. Here to improve readability; could be removed to improve performance
        std::vector<double> influencingSite = {mxTerms[i], myTerms[i], mzTerms[i]};
        if (influencingSite[0] == 0.0 && influencingSite[1] == 0.0 && influencingSite[2] == 0.0) {
            // If influencing site components are all zero, then they don't impact the calculation. So can be skipped
            continue;
        }

        if (exchangeStiffness == 0.0 || _exchangeVec[sitePositions[i]-1] == 0.0) {
            // _exchangeVec[sitePositions[i]-1] refers to exchange vector to the LHS of the current site; [i] is RHS
            continue;
        }

        double latticeConstant = std::sqrt(exchangeStiffness / _exchangeVec[sitePositions[i]-1]);

        if (std::isinf(latticeConstant)) {
            // Guard clause to ensure that the lattice constant is not infinite (backup test / temporary)
            continue;
        }

        std::vector<double> positionVector = {(sitePositions[i] - sitePositions[originIndex]) * latticeConstant, 0, 0};

        double positionVector_norm = positionVector[0];  // Simplifies to this for only a single component

        double positionVector_cubed = std::pow(positionVector_norm, 3);
        double positionVector_fifth = std::pow(positionVector_norm, 5);

        if (positionVector_cubed == 0.0 || positionVector_fifth == 0.0) {
            // Could use an epsilon value here to avoid division by zero and to make the code more efficient
            continue;
        }
        // Calculate the dot products
        double originSiteDotPosition = originSite[0] * positionVector[0];

        double influencingSiteDotPosition = influencingSite[0] * positionVector[0];

        for (int j = 0; j < 3; j++) {
            // Calculate the dipole-dipole coupling term
            double DipoleValue = _dipoleConstant * (((3.0 * positionVector[j] * influencingSiteDotPosition)
                                 / positionVector_fifth) - influencingSite[j] / positionVector_cubed);
            totalDipoleTerms[j] += DipoleValue;
        }
    }

    return totalDipoleTerms;
}
std::vector<double> Numerical_Methods_Class::DipolarInteractionInterlayer(std::vector<std::vector<double>>& mTermsLayer1,
                                                                          std::vector<std::vector<double>>& mTermsLayer2,
                                                                          int& currentSite, const int& currentLayer,
                                                                          const int& otherLayer) {
    std::vector<double> totalDipoleTerms = {0.0, 0.0, 0.0};
    bool findAdj = false;

    double exchangeStiffness = 5.3e-17;
    double interlayerExchange = 132.0;  // Interlayer exchange coupling in Tesla

    if (currentSite <= _numSpinsDamped or currentSite > (_layerSpinsInChain[currentLayer] + _numSpinsDamped)) {
        return {0.0, 0.0, 0.0};  // Ensure currentSite is valid within the current (target) layer
    }

    // Calculate the dipolar coupling for chain1
    std::vector<double> totalDipoleTermsLayer1 = DipolarInteractionIntralayer(mTermsLayer1, currentSite, currentLayer,
                                                                              exchangeStiffness);

    std::vector<double> totalDipoleTermsOtherChains;
    if (findAdj) { totalDipoleTermsOtherChains = DipolarInteractionInterlayerAdjacent(mTermsLayer1, mTermsLayer2,
                                                                                      _numberNeighbours, currentSite,
                                                                                      currentLayer, exchangeStiffness,
                                                                                      interlayerExchange); }
    else { totalDipoleTermsOtherChains = DipolarInteractionInterlayerAll(mTermsLayer1, mTermsLayer2,
                                                                         currentSite, currentLayer, otherLayer,
                                                                         exchangeStiffness, interlayerExchange); }

    // Finally add the three dipole terms to get the total dipole term for a site in chain 1
    for (int i = 0; i < 3; i++) {
        totalDipoleTerms[i] += totalDipoleTermsLayer1[i] + totalDipoleTermsOtherChains[i];
    }

    return totalDipoleTerms;
}
std::vector<double> Numerical_Methods_Class::DipolarInteractionInterlayerAll(std::vector<std::vector<double>>& mTermsLayer1,
                                                                             std::vector<std::vector<double>>& mTermsLayer2,
                                                                             int& currentSite, const int& currentLayer,
                                                                             const int& otherLayer, double& exchangeStiffness,
                                                                             double& interlayerExchange) {
    /* Calculate the dipolar interaction between a site in Layer1 (chain 1), and every other site in another layer (chain 2).
     *
     * WARNING. This function is only valid for the following conditions: the two layers are parallel; the distance
     * between sites in each layer is the same; there is no z-component involved in the position coordinates. The
     * removal of the z-coordinate allows for fewer calculations.
     *
     */

    std::vector<double> totalDipolarInteractionInterlayer = {0.0, 0.0, 0.0};

    // Stop-gap code to prevent memory-access violation error. Needs fixed in the future
    int chainTwoOffset;
    if (!_driveAllLayers) {chainTwoOffset = _layerSpinsInChain[otherLayer] + _numSpinsDamped;}
    else {chainTwoOffset = _layerSpinsInChain[currentLayer] + _numSpinsDamped;}

    for (int otherSite = 0; otherSite < mTermsLayer2.size(); otherSite++) {
        if (otherSite > _numSpinsDamped and otherSite <= chainTwoOffset) {
            // Exclude damped regions as they are aphysical and will lead to incorrect results

            double intralayerLatticeConstant = std::sqrt(exchangeStiffness / _exchangeVec[currentSite]);
            double interlayerLatticeConstant = std::sqrt(exchangeStiffness / interlayerExchange);

            if (std::isinf(intralayerLatticeConstant) or std::isinf(interlayerLatticeConstant)) {
                // Guard clause to ensure that the lattice constant is not infinite (backup test / temporary)
                continue;
            }

            std::vector<double> positionVector = {(otherSite - currentSite) * intralayerLatticeConstant,
                                                  interlayerLatticeConstant, 0};

            double positionVector_norm = std::sqrt(std::pow(positionVector[0], 2) + std::pow(positionVector[1], 2));
            double positionVector_cubed = std::pow(positionVector_norm, 3);
            double positionVector_fifth = std::pow(positionVector_norm, 5);

            std::vector<double> originSite = {mTermsLayer1[currentSite][0] * _muMagnitudeIron,
                                              mTermsLayer1[currentSite][1] * _muMagnitudeIron,
                                              mTermsLayer1[currentSite][2] * _muMagnitudeIron};
            std::vector<double> influencingSite = {mTermsLayer2[otherSite][0] * _muMagnitudeIron,
                                                   mTermsLayer2[otherSite][1] * _muMagnitudeIron,
                                                   mTermsLayer2[otherSite][2] * _muMagnitudeIron};

            double originSiteDotPosition = originSite[0] * positionVector[0] + originSite[1] * positionVector[1];
            double influencingSiteDotPosition = influencingSite[0] * positionVector[0]
                                                + influencingSite[1] * positionVector[1];

            for (int j = 0; j < 3; j++) {
                double DipoleValue = _dipoleConstant * (((3.0 * positionVector[j] * influencingSiteDotPosition)
                                     / positionVector_fifth) - influencingSite[j] / positionVector_cubed);
                totalDipolarInteractionInterlayer[j] += DipoleValue;
            }

        }
    }

    return totalDipolarInteractionInterlayer;
}
std::vector<double> Numerical_Methods_Class::DipolarInteractionInterlayerAdjacent(std::vector<std::vector<double>>& mTermsChain1,
                                                                          std::vector<std::vector<double>>& mTermsChain2,
                                                                          int& numNeighbours, int& currentSite, const int& currentLayer,
                                                                          double& exchangeStiffness, double& interlayerExchange) {
    /* Calculate the dipolar interaction between a site in Layer1 (chain 1), and every other site in another layer
     * (chain 2) within the driving region.
     *
     * WARNING. This function is only valid for the following conditions: the two layers are parallel; the distance
     * between sites in each layer is the same; there is no x- or z-components involved in the position coordinates; the driving
     * region of Layer1 overlaps exactly with the intended dipolar driven region of Layer2.
     *
     * To calculate the dipolar interaction between every site in another chain and your current site, use
     * `DipolarInteractionInterlayerAll`
     *
     */

    std::vector<double> totalDipolarInteractionInterlayer = {0.0, 0.0, 0.0};

    // Stop-gap code to prevent memory-access violation error. Needs fixed in the future
    int chainTwoOffset;
    if (!_driveAllLayers) {chainTwoOffset = _layerSpinsInChain[0] + _numSpinsDamped;}
    else {chainTwoOffset = _layerSpinsInChain[currentLayer] + _numSpinsDamped;}

    // Check if currentSite is a valid index for mTermsChain2 before calculations
    if (currentSite > _numSpinsDamped and currentSite <= chainTwoOffset) {
        // Could also calculate coupling for each site in chain 2, but this is computationally expensive

        double interlayerLatticeConstant = std::sqrt(exchangeStiffness / interlayerExchange);

        std::vector<double> positionVector = {0, interlayerLatticeConstant, 0};
        double positionVector_norm = positionVector[1];  // Simplifies to this for only a single component
        double positionVector_cubed = std::pow(positionVector_norm, 3);
        double positionVector_fifth = std::pow(positionVector_norm, 5);

        std::vector<double> originSite = {mTermsChain1[currentSite][0] * _muMagnitudeIron,
                                          mTermsChain1[currentSite][1] * _muMagnitudeIron,
                                          mTermsChain1[currentSite][2] * _muMagnitudeIron};

        std::vector<double> influencingSite = {mTermsChain2[currentSite][0] * _muMagnitudeIron,
                                               mTermsChain2[currentSite][1] * _muMagnitudeIron,
                                               mTermsChain2[currentSite][2] * _muMagnitudeIron};

        double originSiteDotPosition = originSite[1] * positionVector[1];
        double influencingSiteDotPosition = influencingSite[1] * positionVector[1];

        for (int j = 0; j < 3; j++) {
            double DipoleValue = _dipoleConstant * ((3.0*positionVector[j]*influencingSiteDotPosition) / positionVector_fifth
                                              - influencingSite[j] / positionVector_cubed);
            totalDipolarInteractionInterlayer[j] += DipoleValue;
        }
    }

    return totalDipolarInteractionInterlayer;
}

double Numerical_Methods_Class::GenerateGaussianNoise(const double &mean, const double &stddev) {
    // Function to generate random numbers from a Gaussian distribution
    static std::mt19937 generator(std::random_device{}());
    std::normal_distribution<double> distribution(mean, stddev);
    return distribution(generator);
}
std::vector<double> Numerical_Methods_Class::StochasticTerm(const int& site, const double &timeStep) {
    // Function to compute the stochastic term

    // Compute the standard deviation for the Gaussian noise
    double stddev = std::sqrt(2.0 * _gilbertVector[site] * _boltzmannConstant * _ambientTemperature / (_gyroMagConst * _satMag * timeStep));

    // Generate Gaussian noise for each direction
    double xi_x = GenerateGaussianNoise(0.0, stddev);
    double xi_y = GenerateGaussianNoise(0.0, stddev);
    double xi_z = GenerateGaussianNoise(0.0, stddev);

    return {xi_x, xi_y, xi_z};
}
std::vector<double> Numerical_Methods_Class::ComputeStochasticTerm(const int& site, const double &timeStep) {
    // Function to compute the stochastic term
    std::vector<double> noise = StochasticTerm(site, timeStep);
    std::vector<double> stochasticField = {noise[0], noise[1], noise[2]};
    return stochasticField;
}

__device__ double Numerical_Methods_Class::EffectiveFieldX(const int& site, const int& layer, const double& mxLHS, const double& mxMID,
                                                const double& mxRHS, const double& dipoleTerm, const double& current_time) {
    // The effective field (H_eff) x-component acting upon a given magnetic moment (site), abbreviated to 'hx'
    double hx;

    if (_isFM) {
        if (site >= _drivingRegionLHS && site <= _drivingRegionRHS) {
            // The pulse of input energy will be restricted to being along the x-direction, and it will only be generated within the driving region
            if (_driveAllLayers || layer == 0)
                hx = _exchangeVec[site - 1] * mxLHS + _exchangeVec[site] * mxRHS + dipoleTerm +
                      _dynamicBiasField * cos(_drivingAngFreq * current_time);
            else if  (_hasStaticDrive)
                hx = _exchangeVec[site - 1] * mxLHS + _exchangeVec[site] * mxRHS + dipoleTerm + _dynamicBiasField;
            else if  ((!_driveAllLayers && layer != 0))
                hx = _exchangeVec[site - 1] * mxLHS + _exchangeVec[site] * mxRHS + dipoleTerm;
        } else
            // All spins along x which are not within the driving region
            hx = _exchangeVec[site - 1] * mxLHS + _exchangeVec[site] * mxRHS + dipoleTerm;
    } else if (!_isFM) {
        if (site >= _drivingRegionLHS && site <= _drivingRegionRHS) {
            // The pulse of input energy will be restricted to being along the x-direction, and it will only be generated within the driving region
            if (_hasStaticDrive)
                hx = -1.0 * (_exchangeVec[site - 1] * mxLHS + _exchangeVec[site] * mxRHS + _dynamicBiasField);
            else if (!_hasStaticDrive)
                hx = -1.0 * (_exchangeVec[site - 1] * mxLHS + _exchangeVec[site] * mxRHS) + _dynamicBiasField * cos(_drivingAngFreq * current_time);
        } else
            // All spins along x which are not within the driving region
            hx = -1.0 * (_exchangeVec[site - 1] * mxLHS + _exchangeVec[site] * mxRHS);
    }

    return hx;
}
__device__ double Numerical_Methods_Class::EffectiveFieldY(const int& site, const int& layer, const double& myLHS, const double& myMID, const double& myRHS,
                                                const double &dipoleTerm) {
    // The effective field (H_eff) y-component acting upon a given magnetic moment (site), abbreviated to 'hy'
    double hy;

    if (_isFM) {
        hy = _exchangeVec[site-1] * myLHS + _exchangeVec[site] * myRHS + dipoleTerm;
    } else if (!_isFM) {
        hy = -1.0 * (_exchangeVec[site-1] * myLHS + _exchangeVec[site] * myRHS);
    }

    return hy;
}
__device__ double Numerical_Methods_Class::EffectiveFieldZ(const int& site, const int& layer, const double& mzLHS, const double& mzMID, const double& mzRHS,
                                                const double& dipoleTerm) {
    // The effective field (H_eff) z-component acting upon a given magnetic moment (site), abbreviated to 'hz'
    double hz;

    if (_isFM) {
        hz = _exchangeVec[site-1] * mzLHS + _exchangeVec[site] * mzRHS + dipoleTerm + GV.GetStaticBiasField();
    } else if (!_isFM) {
        if (mzMID > 0)
            hz = GV.GetStaticBiasField() + _anisotropyField - (_exchangeVec[site-1] * mzLHS + _exchangeVec[site] * mzRHS);
        else if (mzMID < 0)
            hz = GV.GetStaticBiasField() - _anisotropyField - (_exchangeVec[site-1] * mzLHS + _exchangeVec[site] * mzRHS);
    }

    return hz;
}

__device__ double Numerical_Methods_Class::MagneticMomentXMulti(const int& site, const int& layer, const double& mxMID, const double& myMID, const double& mzMID,
                                                const double& hxMID, const double& hyMID, const double& hzMID) {

    double mxK;

    if (_useLLG) {
        // The magnetic moment components' coupled equations (obtained from LLG equation) with the parameters for the first stage of RK2.
        mxK = _gyroMagConst * (- (_gilbertVectorMulti[layer][site] * hyMID * mxMID * myMID) + hyMID * mzMID - hzMID * (myMID + _gilbertVectorMulti[layer][site] * mxMID * mzMID) + _gilbertVectorMulti[layer][site] * hxMID * (pow(myMID,2) + pow(mzMID,2)));
    } else {
        // The magnetic moment components' coupled equations (obtained from the torque equation) with the parameters for the first stage of RK2.
        mxK = -1.0 * _gyroMagConst * (myMID * hzMID - mzMID * hyMID);
    }

    return mxK;
}
__device__ double Numerical_Methods_Class::MagneticMomentYMulti(const int& site, const int& layer, const double& mxMID, const double& myMID, const double& mzMID,
                                                const double& hxMID, const double& hyMID, const double& hzMID) {

    double myK;

    if (_useLLG) {
        // The magnetic moment components' coupled equations (obtained from LLG equation) with the parameters for the first stage of RK2.
        myK = _gyroMagConst * (-(hxMID * mzMID) + hzMID * (mxMID - _gilbertVectorMulti[layer][site] * myMID * mzMID) + _gilbertVectorMulti[layer][site] * (hyMID * pow(mxMID,2) - hxMID * mxMID * myMID + hyMID * pow(mzMID,2)));
    } else {
        // The magnetic moment components' coupled equations (obtained from the torque equation) with the parameters for the first stage of RK2.
        myK = _gyroMagConst * (mxMID * hzMID - mzMID * hxMID);
    }

    return myK;
}
__device__ double Numerical_Methods_Class::MagneticMomentZMulti(const int& site, const int& layer, const double& mxMID, const double& myMID, const double& mzMID,
                                                const double& hxMID, const double& hyMID, const double& hzMID) {

    double mzK;

    if (_useLLG) {
        // The magnetic moment components' coupled equations (obtained from LLG equation) with the parameters for the first stage of RK2.
        mzK = _gyroMagConst * (hxMID * myMID + _gilbertVectorMulti[layer][site] * hzMID * (pow(mxMID,2) + pow(myMID,2)) - _gilbertVectorMulti[layer][site]*hxMID*mxMID*mzMID - hyMID * (mxMID + _gilbertVectorMulti[layer][site] * myMID * mzMID));
    } else {
        // The magnetic moment components' coupled equations (obtained from the torque equation) with the parameters for the first stage of RK2.
        mzK = -1.0 * _gyroMagConst * (mxMID * hyMID - myMID * hxMID);
    }

    return mzK;
}

void Numerical_Methods_Class::SolveRK2() {
    // Uses multiple layers to solve the RK2 midpoint method. See the documentation for more details.

    // Create files to save the data. All files will have (GV.GetFileNameBase()) in them to make them clearly identifiable.
    std::ofstream mxRK2File(GV.GetFilePath() + "rk2_mx_" + GV.GetFileNameBase() + ".csv");
    std::ofstream mxRK2File1;
    if (_useMultilayer) { std::ofstream mxRK2File1(GV.GetFilePath() + "rk2_mx1_" + GV.GetFileNameBase() + ".csv"); }

    // User information and file header is magnetic-material specific.
    if (_isFM) {
        InformUserOfCodeType("RK2 Midpoint (FM)");
        CreateFileHeader(mxRK2File, "RK2 Midpoint (FM)", false, 0);
        if (_useMultilayer) { CreateFileHeader(mxRK2File1, "RK2 Midpoint (FM)", false, 1); }
    } else if (!_isFM) {
        InformUserOfCodeType("RK2 Midpoint (AFM)");
        CreateFileHeader(mxRK2File, "RK2 Midpoint (AFM)");
        if (_useMultilayer)  { CreateFileHeader(mxRK2File1, "RK2 Midpoint (AFM)"); }
    }

    if (GV.GetEmailWhenCompleted()) {
        CreateMetadata();
    }

    progressbar bar(100);

    // Nested vectors are that allow for multiple layers to be used in the code. See documentation for more details.
    double zeroValue = 0.0;
    std::vector<std::vector<std::vector<double>>> m0Nest = InitialiseNestedVectors(_totalLayers, _mxInit, _myInit, _mzInit);
    std::vector<std::vector<std::vector<double>>> m1Nest = InitialiseNestedVectors(_totalLayers, _mxInit, _myInit, zeroValue);
    std::vector<std::vector<std::vector<double>>> m2Nest = InitialiseNestedVectors(_totalLayers, _mxInit, _myInit, zeroValue);

    for (int iteration = _iterationStart; iteration <= _iterationEnd; iteration++) {

        if (_iterationEnd >= 100 && iteration % (_iterationEnd / 100) == 0)
            // Doesn't work on Windows due to different compiler. Doesn't work for fewer than 100 iterations
            bar.update();

        TestShockwaveConditions(iteration);

        double t0 = _totalTime, t0HalfStep = _totalTime + _stepsizeHalf;

        double* m0Flat = flattenNestedVector(m0Nest[0]);
        double* m1Flat = flattenNestedVector(m1Nest[0]);

        // Can improve this by using my private _ variables later
        int m0FlatNumElements = m0Nest[0].size() * m0Nest[0][0].size();
        int m1FlatNumElements = m1Nest[0].size() * m1Nest[0][0].size();

        RK2Stage1CUDA(0, t0, m0Flat, m0FlatNumElements);
        RK2Stage2CUDA(0, t0HalfStep, m1Flat, m1FlatNumElements);

        // Need to assign each element of the flattened array to the correct element of the nested vector.
        // This is still to be done

        // Free the memory used
        delete[] m0Flat;
        delete[] m1Flat;

        // Everything below here is part of the class method, but not the internal RK2 stage loops.

        /**
         * Removes (possibly) large arrays as they can lead to memory overloads later in main.cpp. Failing to clear
         * these between loop iterations sometimes led to incorrect values cropping up.
         */

        SaveDataToFileMultilayer(mxRK2File, m2Nest[0], iteration, 0);
        if (_useMultilayer)  { SaveDataToFileMultilayer(mxRK2File1, m2Nest[1], iteration, 1); }

        //Sets the final value of the current iteration of the loop to be the starting value of the next loop.
        m0Nest = m2Nest;

        if (iteration == _forceStopAtIteration)
            exit(0);

        _totalTime += _stepsize;
    }// Final line of RK2 solver for all iterations. Everything below here occurs after RK2 method is complete

    // Ensures files are closed; sometimes are left open if the writing process above fails
    mxRK2File.close();
    if (_useMultilayer) { mxRK2File1.close(); }

    if (GV.GetEmailWhenCompleted()) {
        CreateMetadata(true);
    }

    if (_shouldTrackMValues) {
        std::cout << "\nMax norm. values of M are: ";
        for (int i = 0; i < _largestMNormMulti.size(); i++) {
            if (_largestMNormMulti[i] > 1e-50) { std::cout << "Layer " << i << ": " << _largestMNormMulti[i] << " | "; }
        }
    }

    // Filename can be copy/pasted from C++ console to Python function's console.
    std::cout << "\n\nFile can be found at:\n\t" << GV.GetFilePath() << GV.GetFileNameBase() << std::endl;
}

void Numerical_Methods_Class::RK2Stage1CUDA(int layer, double timeStep, double* m0NestFlattened, int m0FlatDim) {
    // Initialize CUDA
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount == 0) {
        std::cerr << "There's no CUDA-compatible device" << std::endl;
        return;
    }
    hipSetDevice(0);

    // Allocate device memory
    int* d_layerTotalSpins;
    hipMalloc((void**)&d_layerTotalSpins, _layerTotalSpins[layer] * sizeof(int));

    double* d_m0Nest;
    hipMalloc((void**)&d_m0Nest, m0FlatDim * sizeof(double));

    // Copy vectors from Host (CPU) to Device (GPU)
    hipMemcpy(d_layerTotalSpins, _layerTotalSpins[layer], _layerTotalSpins[layer] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_m0Nest, m0NestFlattened, m0FlatDim * sizeof(double), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    RK2KernelStage1<<<_totalLayers,*max_element(_layerTotalSpins.begin(), _layerTotalSpins.end())>>>(d_layerTotalSpins, _totalLayers, d_m0Nest, timeStep, m0FlatDim);

    // Copy results from Device (GPU) to Host (CPU)
    hipMemcpy(_layerTotalSpins, d_layerTotalSpins, _layerTotalSpins[layer] * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(m0NestFlattened, d_m0Nest, m0FlatDim * sizeof(double), hipMemcpyDeviceToHost);


    // Clean up
    hipFree(d_layerTotalSpins);
    hipFree(d_m0Nest);
}
void Numerical_Methods_Class::RK2Stage2CUDA(int layer, double timeStep, double* m1NestFlattened, int m1FlatDim) {
    // Initialize CUDA
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount == 0) {
        std::cerr << "There's no CUDA-compatible device" << std::endl;
        return;
    }
    hipSetDevice(0);

    // Allocate device memory
    int* d_layerTotalSpins;
    hipMalloc((void**)&d_layerTotalSpins, _layerTotalSpins[layer] * sizeof(int));

    double* d_m1Nest;
    hipMalloc((void**)&d_m1Nest, m1FlatDim * sizeof(double));

    // Copy vectors from Host (CPU) to Device (GPU)
    hipMemcpy(d_layerTotalSpins, _layerTotalSpins[layer], _layerTotalSpins[layer] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_m1Nest, m1NestFlattened, m1FlatDim * sizeof(double), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    RK2KernelStage2<<<_totalLayers,*max_element(_layerTotalSpins.begin(), _layerTotalSpins.end())>>>(d_layerTotalSpins, _totalLayers, d_m1Nest, timeStep, m1FlatDim);

    // Copy results from Device (GPU) to Host (CPU)
    hipMemcpy(_layerTotalSpins.data(), d_layerTotalSpins, _layerTotalSpins[layer] * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(m1NestFlattened.data(), d_m1Nest, m1FlatDim * sizeof(double), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_layerTotalSpins);
    hipFree(d_m1Nest);
}

__global__ void RK2KernelStage1(int* d_layerTotalSpins, int totalLayers) {
    // CUDA kernel function
    int layer = blockIdx.x;
    int site = threadIdx.x;

    for (int layer = 0; layer < _totalLayers; layer++) {
        // RK2 Stage 1. Takes initial conditions as inputs.

        for (int site = 1; site <= _layerTotalSpins[layer]; site++) {
            // Exclude the 0th and last spins as they will always be zero-valued (end, pinned, bound spins)

            // Relative to the current site (site); site to the left (LHS); site to the right (RHS)
            int spinLHS = site - 1, spinRHS = site + 1;

            double mxLHS = m0Nest[layer][spinLHS][0], mxMID = m0Nest[layer][site][0], mxRHS = m0Nest[layer][spinRHS][0];
            double myLHS = m0Nest[layer][spinLHS][1], myMID = m0Nest[layer][site][1], myRHS = m0Nest[layer][spinRHS][1];
            double mzLHS = m0Nest[layer][spinLHS][2], mzMID = m0Nest[layer][site][2], mzRHS = m0Nest[layer][spinRHS][2];

            double dipoleX, dipoleY, dipoleZ;
            if (_useDipolar) {

                int layer1, layer2;
                if (layer == 0) {layer1 = 0; layer2 = 1;}
                else if (layer == 1) {layer1 = 1; layer2 = 0;}

                if (_debugFunc) {std::cout << "\n\niteration: " << iteration << " | layer: " << layer << " | site: " << site << std::endl;}
                std::vector<double> dipoleTerms = DipolarInteractionInterlayer(m0Nest[layer1], m0Nest[layer2], site,
                                                                               layer1, layer2);

                dipoleX = dipoleTerms[0];
                dipoleY = dipoleTerms[1];
                dipoleZ = dipoleTerms[2];
            } else {
                dipoleX = 0;
                dipoleY = 0;
                dipoleZ = 0;
            }

            // Calculations for the effective field (H_eff), coded as symbol 'h', components of the target site
            double hxK0 = EffectiveFieldX(site, layer, mxLHS, mxMID, mxRHS, dipoleX, t0);
            double hyK0 = EffectiveFieldY(site, layer, myLHS, myMID, myRHS, dipoleY);
            double hzK0 = EffectiveFieldZ(site, layer, mzLHS, mzMID, mzRHS, dipoleZ);

            // RK2 K-value calculations for the magnetic moment, coded as symbol 'm', components of the target site
            double mxK1, myK1, mzK1;
            if (_useMultilayer) {
                mxK1 = MagneticMomentXMulti(site, layer, mxMID, myMID, mzMID, hxK0, hyK0, hzK0);
                myK1 = MagneticMomentYMulti(site, layer, mxMID, myMID, mzMID, hxK0, hyK0, hzK0);
                mzK1 = MagneticMomentZMulti(site, layer, mxMID, myMID, mzMID, hxK0, hyK0, hzK0);
            } else {
                mxK1 = MagneticMomentX(site, mxMID, myMID, mzMID, hxK0, hyK0, hzK0);
                myK1 = MagneticMomentY(site, mxMID, myMID, mzMID, hxK0, hyK0, hzK0);
                mzK1 = MagneticMomentZ(site, mxMID, myMID, mzMID, hxK0, hyK0, hzK0);
            }

            // Find (m0 + k1/2) for each site, which is used in the next stage.
            m1Nest[layer][site][0] = mxMID + _stepsizeHalf * mxK1;
            m1Nest[layer][site][1] = myMID + _stepsizeHalf * myK1;
            m1Nest[layer][site][2] = mzMID + _stepsizeHalf * mzK1;
        }
    }
}
__global__ void RK2KernelStage2(int* d_layerTotalSpins, int totalLayers) {
    // CUDA kernel function
    int layer = blockIdx.x;
    int site = threadIdx.x;

    for (int layer = 0; layer < _totalLayers; layer++) {
        // RK2 Stage 2. Takes (m0 + k1/2) as inputs.
        for (int site = 1; site <= _layerTotalSpins[layer]; site++) {

            // Relative to the current site (site); site to the left (LHS); site to the right (RHS)
            int spinLHS = site - 1, spinRHS = site + 1;

            double mxLHS = m1Nest[layer][spinLHS][0], mxMID = m1Nest[layer][site][0], mxRHS = m1Nest[layer][spinRHS][0];
            double myLHS = m1Nest[layer][spinLHS][1], myMID = m1Nest[layer][site][1], myRHS = m1Nest[layer][spinRHS][1];
            double mzLHS = m1Nest[layer][spinLHS][2], mzMID = m1Nest[layer][site][2], mzRHS = m1Nest[layer][spinRHS][2];

            double dipoleX, dipoleY, dipoleZ;
            if (_useDipolar) {

                int layer1, layer2;
                if (layer == 0) {layer1 = 0; layer2 = 1;}
                else if (layer == 1) {layer1 = 1; layer2 = 0;}

                int debugCounter = 0;  // To make sure debug outputs only occur during the first RK2 stage, not this second stage
                if (_debugFunc) { _debugFunc = false; debugCounter++; }
                std::vector<double> dipoleTerms = DipolarInteractionInterlayer(m1Nest[layer1], m1Nest[layer2], site,
                                                                               layer1, layer2);
                if (debugCounter > 0) { _debugFunc = true; }

                dipoleX = dipoleTerms[0];
                dipoleY = dipoleTerms[1];
                dipoleZ = dipoleTerms[2];
            } else {
                dipoleX = 0;
                dipoleY = 0;
                dipoleZ = 0;
            }
            // Calculations for the effective field (H_eff), coded as symbol 'h', components of the target site
            double hxK1 = EffectiveFieldX(site, layer, mxLHS, mxMID, mxRHS, dipoleX, t0);
            double hyK1 = EffectiveFieldY(site, layer, myLHS, myMID, myRHS, dipoleY);
            double hzK1 = EffectiveFieldZ(site, layer, mzLHS, mzMID, mzRHS, dipoleZ);

            // RK2 K-value calculations for the magnetic moment, coded as symbol 'm', components of the target site
            double mxK2, myK2, mzK2;
            if (_useMultilayer) {
                mxK2 = MagneticMomentXMulti(site, layer, mxMID, myMID, mzMID, hxK1, hyK1, hzK1);
                myK2 = MagneticMomentYMulti(site, layer, mxMID, myMID, mzMID, hxK1, hyK1, hzK1);
                mzK2 = MagneticMomentZMulti(site, layer, mxMID, myMID, mzMID, hxK1, hyK1, hzK1);
            } else {
                mxK2 = MagneticMomentX(site, mxMID, myMID, mzMID, hxK1, hyK1, hzK1);
                myK2 = MagneticMomentY(site, mxMID, myMID, mzMID, hxK1, hyK1, hzK1);
                mzK2 = MagneticMomentZ(site, mxMID, myMID, mzMID, hxK1, hyK1, hzK1);
            }

            m2Nest[layer][site][0] = m0Nest[layer][site][0] + _stepsize * mxK2;
            m2Nest[layer][site][1] = m0Nest[layer][site][1] + _stepsize * myK2;
            m2Nest[layer][site][2] = m0Nest[layer][site][2] + _stepsize * mzK2;

            if (_shouldTrackMValues) {
                double mIterationNorm = sqrt(
                        pow(m2Nest[layer][site][0], 2) + pow(m2Nest[layer][site][1], 2) + pow(m2Nest[layer][site][2], 2));
                if ((_largestMNormMulti[layer]) > (1.0 - mIterationNorm)) { _largestMNormMulti[layer] = (1.0 - mIterationNorm); }
            }
        }
    }
}

void Numerical_Methods_Class::InformUserOfCodeType(const std::string& nameNumericalMethod) {
    /**
     * Informs the user of the code type they are running, including: solver type; special modules.
     */
    if (_useLLG)
        std::cout << "\nYou are running the " << nameNumericalMethod << " Spinchains (LLG) code";
    else
        std::cout << "\nYou are running the " << nameNumericalMethod << " Spinchains (Torque) code";

    if (_hasShockwave)
        std::cout << " with shockwave module.\n";
    else
        std::cout << ".\n";

}
void Numerical_Methods_Class::TestShockwaveConditions(double iteration) {

    if (_shouldDriveCease) {
        // and (_isShockwaveOn and _isShockwaveAtMax)) {
        if (_isShockwaveOn and not _isShockwaveAtMax) {
            std::cout << "Shock not at maximum when cut-off" << std::endl;
        }

        if (iteration >= _iterationEnd * _iterEndShock) {
            // Shockwave begins once simulation is a certain % complete
            _hasShockwave = false;
            _isShockwaveOn = false;
            _dynamicBiasField = 0;
        }

        return;

    }

    // If method is triggered, then the applied biasFieldDriving is increased by the scale factor _shockwaveScaling
    if (_hasShockwave and not _isShockwaveOn)
    {
        if (iteration >= _iterationEnd * _iterStartShock)
        {
            // Shockwave begins once simulation is a certain % complete
            _isShockwaveOn = true;
            _dynamicBiasField = _shockwaveInitialStrength;
        }

        return;
    }

    if (_isShockwaveOn and not _isShockwaveAtMax)
    {
        _dynamicBiasField += _shockwaveStepsize;

        if (_dynamicBiasField >= _shockwaveMax)
        {
            _dynamicBiasField = _shockwaveMax;
            _isShockwaveAtMax = true;

        }
        return;

    }

}
void Numerical_Methods_Class::CreateMetadata(bool print_end_time) {

    std::string file_name = "simulation_metadata.txt";

    if (print_end_time) {
        std::ofstream metadata_end;
        metadata_end.open(GV.GetFilePath() + file_name, std::ios_base::app); // append instead of overwrite
        auto end = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
        metadata_end << "Finished at:\t" << std::put_time(localtime(&end), "%F %H-%M-%S") << std::endl;
        metadata_end.close();
    }
    else {
        std::ofstream metadata_start(GV.GetFilePath() + file_name);
        CreateFileHeader(metadata_start, "NM 2", true);
        auto start = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
        metadata_start << "Started at:\t" << std::put_time(localtime(&start), "%F %H-%M-%S") << std::endl;
        metadata_start.close();
    }
}
void Numerical_Methods_Class::CreateFileHeader(std::ofstream &outputFileName, std::string methodUsed, bool is_metadata, int layer) {
    /**
     * Write all non-data information to the output file.
     */
    if (is_metadata) {
        outputFileName << "Key Data\n\n";

        outputFileName << "[Booleans where (1) indicates (True) and (0) indicates (False)]\n";

        outputFileName << "Using LLG: [" << _useLLG << "]\t\t\t\tUsing Shockwave: [" << _hasShockwave << "]\t\tDrive from LHS: [" << _lhsDrive <<
                       "]\nNumerical Method Used: [" << methodUsed << "]\t\tHas Static Drive: [" << _hasStaticDrive << "]\n";

        outputFileName << "\n";

        outputFileName << "Static Bias Field (H0): " << GV.GetStaticBiasField() << " T\t\t\t" << "Dynamic Bias Field (H_D1): " << _dynamicBiasField << " T\n" <<
                          "Dynamic Bias Field Scale Factor: " << _shockwaveInitialStrength << "\t\t" << "Second Dynamic Bias Field (H_D2): " << _shockwaveMax << " T\n" <<
                          "Driving Frequency (f): " << _drivingFreq << "Hz\t\t""Driving Region Start Site: " << _drivingRegionLHS - _numSpinsDamped << "\n" <<
                          "Driving Region End Site: " << _drivingRegionRHS - _numSpinsDamped << " \t\t\t" << "Driving Region Width: " << _drivingRegionWidth << " \n" <<
                          "Max. Sim. Time: " << _maxSimTime << " s\t\t\t\t" << "Min. Exchange Val (J): " << GV.GetExchangeMinVal()  << " T\n" <<
                          "Max. Exchange Val (J): " << GV.GetExchangeMaxVal() << " T\t\t\t" << "Max. Iterations: " << _iterationEnd << "\n" <<
                          "No. DataPoints: " << _numberOfDataPoints << " \t\t\t\t" << "No. Spins in Chain: " << _layerSpinsInChain[layer] << "\n" <<
                          "No. Damped Spins: " << _numSpinsDamped << "per side\t\t\t" << "No. Total Spins: " << _layerTotalSpins[layer] << " \n" <<
                          "Stepsize (h): " << _stepsize << "\t\t\t\t" << "Gilbert Damping Factor: " << _gilbertConst << "\n" <<
                          "Gyromagnetic Ratio (2Pi*Y): " << _gyroMagConst << "\t\t""Shockwave Gradient Time: " << _iterStartShock << "s\n" <<
                          "Shockwave Application Time: " << _shockwaveGradientTime * _stepsize << "s\n" <<
                          std::endl;

        return;
    }
    else {

        outputFileName << "Key Data\n";

        outputFileName << "[Booleans where (1) indicates (True) and (0) indicates (False)]\n";

        outputFileName << "Using LLG," << _useLLG << ",Using Shockwave," << _hasShockwave << ",Drive from LHS," << _lhsDrive <<
                       ",Numerical Method Used," << methodUsed << ",Has Static Drive," << _hasStaticDrive << "\n";

        outputFileName << "\n";

        outputFileName << "Static Bias Field (H0) [T],Dynamic Bias Field (H_D1) [T],Dynamic Bias Field Scale Factor,Second Dynamic Bias Field (H_D2)[T],"
                          "Driving Frequency (f) [Hz],Driving Region Start Site,Driving Region End Site, Driving Region Width,"
                          "Max. Sim. Time [s],Min. Exchange Val (J)[T],Max. Exchange Val (J)[T],Max. Iterations,No. DataPoints,"
                          "No. Spins in Chain (N),No. Damped Spins (per side),No. Total Spins, Stepsize (h),Gilbert Damping Factor, Gyromagnetic Ratio (2Pi*Y),"
                          "Shockwave Gradient Time [s], Shockwave Application Time [s]"
                          "\n";

        outputFileName << GV.GetStaticBiasField() << ", " << _dynamicBiasField << ", " << _shockwaveInitialStrength << ", " << _shockwaveMax << ", "
                       << _drivingFreq << ", " << _drivingRegionLHS - _numSpinsDamped << ", " << _drivingRegionRHS - _numSpinsDamped << ", " << _drivingRegionWidth << ", "
                       << _maxSimTime << ", " << GV.GetExchangeMinVal() << ", " << GV.GetExchangeMaxVal() << ", " << _iterationEnd << ", " << _numberOfDataPoints << ", "
                       << _layerSpinsInChain[layer] << ", " << _numSpinsDamped << ", " << _layerTotalSpins[layer] << ", " << _stepsize << ", " << _gilbertConst << ", " << _gyroMagConst << ", "
                       << _iterStartShock << ", " << _shockwaveGradientTime * _stepsize
                       << "\n";

        outputFileName << "\n";
    }

    std::string notesComments;
    std::cout << "Enter any notes for this simulation: ";
    std::cin.ignore();
    std::getline(std::cin, notesComments );
    outputFileName << "Note(s):," << notesComments << "\n"; // Adding comma ensures the note itself is in a different csv cell to the term 'Note(s):'

    outputFileName << "[Column heading indicates the spin site (#) being recorded. Data is for the (mx) component]\n";

    outputFileName << "\n";

    CreateColumnHeaders(outputFileName, layer);

    std::cout << "\n";
}
void Numerical_Methods_Class::CreateColumnHeaders(std::ofstream &outputFileName, int& layer) {
    /**
     * Creates the column headers for each spin site simulated. This code can change often, so compartmentalising it in
     * a separate function is necessary to reduce bugs.
     */
    if (_printAllData or _printFixedLines) {
        // Print column heading for every spin simulated.
        outputFileName << "Time [s], ";
        for (int i = 1; i <= _layerTotalSpins[layer]; i++) {
            outputFileName << i << ", ";
        }
        outputFileName << std::endl;

    } else if (_printFixedSites) {

        outputFileName << "Time";
        for (int & fixed_out_val : _fixed_output_sites)
            outputFileName << "," << fixed_out_val;
        outputFileName << std::endl;

        //outputFileName << "Time" << ", "
        //               << static_cast<int>(14000) << ","
        //               << static_cast<int>(16000) << ","
        //               << static_cast<int>(18000) << ","
        //               << static_cast<int>(20000) << std::endl;

    }
}
double* Numerical_Methods_Class::flattenNestedVector(const std::vector<std::vector<double>>& nestedVector) {
    int numElements = nestedVector.size() * nestedVector[0].size();
    double* flattenedArray = new double[3 * numElements];

    for (int i = 0; i < nestedVector.size(); ++i) {
        for (int j = 0; j < nestedVector[i].size(); ++j) {
            flattenedArray[j * nestedVector.size() + i] = nestedVector[i][j];
        }
    }

    return flattenedArray;
}
void Numerical_Methods_Class::SaveDataToFileMultilayer(std::ofstream &outputFileName, std::vector<std::vector<double>> &nestedArrayToWrite, int &iteration, int layer) {
    std::cout.precision(6);
    std::cout << std::scientific;

    std::vector<double> arrayToWrite;
    // Extract the first element from each nested vector
    for (const auto& innerVector : nestedArrayToWrite) {
        if (!innerVector.empty()) {
            arrayToWrite.push_back(innerVector[0]);
        }
    }

    if (iteration % (_iterationEnd / _numberOfDataPoints) == 0) {
        if (_printFixedLines) {
            for (int i = 0; i <= _layerTotalSpins[layer]; i++) {
                // Steps through vectors containing all mag. moment components and saves to files
                if (i == 0)
                    // Print current time
                    outputFileName << (iteration * _stepsize) << ",";

                else if (i == _layerTotalSpins[layer])
                    // Ensures that the final line doesn't contain a comma.
                    outputFileName << arrayToWrite[i] << std::flush;

                else
                    // For non-special values, write the data.
                    outputFileName << arrayToWrite[i] << ", ";
            }
            // Take new line after current row is finished being written.
            outputFileName << std::endl;

            return;
        } else if (_printFixedSites) {
            /*outputFileName << (iteration * _stepsize) << ","
               << arrayToWrite[14000] << ","
               << arrayToWrite[16000] << ","
               << arrayToWrite[18000] << ","
               << arrayToWrite[20000] << std::endl;
               */
            outputFileName << (iteration * _stepsize);
            for (int & fixed_out_val : _fixed_output_sites)
                outputFileName << "," << arrayToWrite[fixed_out_val];
            outputFileName << std::endl;

            return;
        }
    }

    if (_printAllData) {
        for (int i = 0; i <= _layerTotalSpins[layer]; i++) {
            // Steps through vectors containing all mag. moment components found at the end of RK2-Stage 2, and saves to files
            if (i == 0)
                outputFileName << (iteration * _stepsize) << ","; // Print current time
            else if (i == _layerTotalSpins[layer])
                outputFileName << arrayToWrite[i] << std::flush; // Ensures that the final line doesn't contain a comma.
            else
                outputFileName << arrayToWrite[i] << ","; // For non-special values, write the data.
        }
        outputFileName << std::endl;

        return;
    }

    /*
    if (_printFixedLines) {
        // iteration >= static_cast<int>(_iterationEnd / 2.0) &&
        if (iteration % (_iterationEnd / _numberOfDataPoints) == 0) {
            //if (iteration == _iterationEnd) {
            for (int i = 0; i <= GV.GetNumSpins(); i++) {
                // Steps through vectors containing all mag. moment components and saves to files
                if (i == 0)
                    // Print current time
                    outputFileName << (iteration * _stepsize) << ",";

                else if (i == GV.GetNumSpins())
                    // Ensures that the final line doesn't contain a comma.
                    outputFileName << arrayToWrite[i] << std::flush;

                else
                    // For non-special values, write the data.
                    outputFileName << arrayToWrite[i] << ", ";
            }
            // Take new line after current row is finished being written.
            outputFileName << std::endl;
        }
    } else {
        if (_printAllData) {
            for (int i = 0; i <= GV.GetNumSpins(); i++) {
                // Steps through vectors containing all mag. moment components found at the end of RK2-Stage 2, and saves to files
                if (i == 0)
                    outputFileName << (iteration * _stepsize) << ","; // Print current time
                else if (i == GV.GetNumSpins())
                    outputFileName << arrayToWrite[i] << std::flush; // Ensures that the final line doesn't contain a comma.
                else
                    outputFileName << arrayToWrite[i] << ","; // For non-special values, write the data.
            }
            outputFileName << std::endl; // Take new line after current row is finished being written.
        } else {
            if (iteration % (_iterationEnd / _numberOfDataPoints) == 0) {
                if (_printFixedSites) {

                    outputFileName << (iteration * _stepsize) << ","
                                   << arrayToWrite[_drivingRegionLHS] << ","
                                   << arrayToWrite[static_cast<int>(_drivingRegionWidth / 2.0)] << ","
                                   << arrayToWrite[_drivingRegionRHS] << ","
                                   << arrayToWrite[static_cast<int>(1500)] << ","
                                   << arrayToWrite[static_cast<int>(2500)] << ","
                                   << arrayToWrite[static_cast<int>(3500)] << ","
                                   << arrayToWrite[GV.GetNumSpins()] << std::endl;

                    outputFileName << (iteration * _stepsize) << ","
                                   << arrayToWrite[400] << ","
                                   << arrayToWrite[1500] << ","
                                   << arrayToWrite[3000] << ","
                                   << arrayToWrite[4500] << ","
                                   << arrayToWrite[5600] << std::endl;
                } else {
                    outputFileName << (iteration * _stepsize) << ","
                                   << arrayToWrite[_drivingRegionLHS] << ","
                                   << arrayToWrite[static_cast<int>(_drivingRegionWidth / 2.0)] << ","
                                   << arrayToWrite[_drivingRegionRHS] << ","
                                   << arrayToWrite[static_cast<int>(GV.GetNumSpins() / 4.0)] << ","
                                   << arrayToWrite[static_cast<int>(GV.GetNumSpins() / 2.0)] << ","
                                   << arrayToWrite[3 * static_cast<int>(GV.GetNumSpins() / 4.0)] << ","
                                   << arrayToWrite[GV.GetNumSpins()] << std::endl;
                }
            }
        }
    } */
}

